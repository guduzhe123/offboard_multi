#include "hip/hip_runtime.h"
//
// Created by zhouhua on 2021/4/28.
//

#include "motion_plan/poly_traj/use_cuda.cuh"

void print_matrix(int R, int C, double* A, const char* name);

template<typename T>
void PrintEMatrix(const T &mat, const char *name) {
    std::cout << name << " =\n";
    std::cout << mat << std::endl;
}
//##############################################################################
template<typename T>
__global__
void Ker_CopyUpperSubmatrix(const T *__restrict d_in,
                            T *__restrict d_ou,
                            const int M, const int N, const int subM) {
    const int i = threadIdx.x + blockIdx.x*blockDim.x;
    const int j = threadIdx.y + blockIdx.y*blockDim.y;
    if (i>=subM || j>=N)
        return;
    d_ou[j*subM+i] = d_in[j*M+i];
}
//##############################################################################

void UseCuda::onInit() {

#define CUSOLVER_ERRCHK(x) \
        assert((x) == HIPSOLVER_STATUS_SUCCESS && "cusolver failed");
#define CUBLAS_ERRCHK(x) \
        assert((x) == HIPBLAS_STATUS_SUCCESS && "cublas failed");
    CUSOLVER_ERRCHK(hipsolverDnCreate(&cusolverH));
    CUBLAS_ERRCHK(hipblasCreate(&cublasH));
}

void UseCuda::calMatrixInverse(const Eigen::MatrixXd &A, Eigen::MatrixXd &x_sol, float &use_time) {
    typedef double T; // NOTE: don't change this. blas has different func name
    typedef Eigen::Matrix<T,Eigen::Dynamic,Eigen::Dynamic> MatrixXd;

    // define handles
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;

    const int M = A.rows();
    const int N = A.cols();

    if (M != N) return;
    const int K = M;

    x_sol.resize(N,K);
//    x_ref = A.inverse();
    MatrixXd b = MatrixXd::Identity(N,K);

    float elapsed_time;
    hipEvent_t start1, stop1;
    CHECK(hipEventCreate(&start1));
    CHECK(hipEventCreate(&stop1));
    CHECK(hipEventRecord(start1));
    hipEventQuery(start1);
/*    PrintEMatrix(A, "A");
    PrintEMatrix(b, "b");
/*    std::cout << "solution l1 error = " << (x_ref - C).norm()
              << std::endl;*/

#define CUSOLVER_ERRCHK(x) \
        assert((x) == HIPSOLVER_STATUS_SUCCESS && "cusolver failed");
#define CUBLAS_ERRCHK(x) \
        assert((x) == HIPBLAS_STATUS_SUCCESS && "cublas failed");

    CUSOLVER_ERRCHK(hipsolverDnCreate(&cusolverH));
    CUBLAS_ERRCHK(hipblasCreate(&cublasH));



    T *d_A, *d_b, *d_work, *d_work2, *d_tau;
    int *d_devInfo, devInfo;
    CHECK(hipMalloc((void **) &d_A, sizeof(T) * M * N));
    CHECK(hipMalloc((void **) &d_b, sizeof(T) * M * K));
    CHECK(hipMalloc((void **) &d_tau, sizeof(T) * M));
    CHECK(hipMalloc((void **) &d_devInfo, sizeof(int)));
    CHECK(hipMemcpy(d_A, A.data(), sizeof(T) * M * N, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b.data(), sizeof(T) * M * K, hipMemcpyHostToDevice));
    int bufSize, bufSize2;

    // in-place A = QR
    CUSOLVER_ERRCHK(
            hipsolverDnDgeqrf_bufferSize(
                    cusolverH,
                    M,
                    N,
                    d_A,
                    M,
                    &bufSize
            )
    );
    CHECK(hipMalloc((void **) &d_work, sizeof(T) * bufSize));
    CUSOLVER_ERRCHK(
            hipsolverDnDgeqrf(
                    cusolverH,
                    M,
                    N,
                    d_A,
                    M,
                    d_tau,
                    d_work,
                    bufSize,
                    d_devInfo
            )
    );
    CHECK(hipMemcpy(&devInfo, d_devInfo, sizeof(int),
                     hipMemcpyDeviceToHost));
    assert(0 == devInfo && "QR factorization failed");

    // Q^T*b
    CUSOLVER_ERRCHK(
            hipsolverDnDormqr_bufferSize(
                    cusolverH,
                    HIPBLAS_SIDE_LEFT,
                    HIPBLAS_OP_T,
                    M,
                    K,
                    N,
                    d_A,
                    M,
                    d_tau,
                    d_b,
                    M,
                    &bufSize2
            )
    );
    CHECK(hipMalloc((void **) &d_work2, sizeof(T) * bufSize2));
    CUSOLVER_ERRCHK(
            hipsolverDnDormqr(
                    cusolverH,
                    HIPBLAS_SIDE_LEFT,
                    HIPBLAS_OP_T,
                    M,
                    K,
                    min(M, N),
                    d_A,
                    M,
                    d_tau,
                    d_b,
                    M,
                    d_work2,
                    bufSize2,
                    d_devInfo
            )
    );
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(&devInfo, d_devInfo, sizeof(int),
                     hipMemcpyDeviceToHost));
    assert(0 == devInfo && "Q^T b failed");

    // need to explicitly copy submatrix for the triangular solve
    T *d_R, *d_b_;
    CHECK(hipMalloc((void **) &d_R, sizeof(T) * N * N));
    CHECK(hipMalloc((void **) &d_b_, sizeof(T) * N * K));
    dim3 thd_size(10, 10);
    dim3 blk_size((N + thd_size.x - 1) / thd_size.x, (N + thd_size.y - 1) / thd_size.y);
    Ker_CopyUpperSubmatrix<T><<<blk_size, thd_size>>>(d_A, d_R, M, N, N);
    blk_size = dim3((N + thd_size.x - 1) / thd_size.x, (K + thd_size.y - 1) / thd_size.y);
    Ker_CopyUpperSubmatrix<T><<<blk_size, thd_size>>>(d_b, d_b_, M, K, N);

    // solve x = R \ (Q^T*B)
    const double one = 1.0;
    CUBLAS_ERRCHK(
            hipblasDtrsm(
                    cublasH,
                    HIPBLAS_SIDE_LEFT,
                    HIPBLAS_FILL_MODE_UPPER,
                    HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT,
                    N,
                    K,
                    &one,
                    d_R,
                    N,
                    d_b_,
                    N
            )
    );
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(x_sol.data(), d_b_, sizeof(T) * N * K,
                     hipMemcpyDeviceToHost));

    CHECK(hipEventRecord(stop1));
    CHECK(hipEventSynchronize(stop1));
    CHECK(hipEventElapsedTime(&elapsed_time, start1, stop1));
    use_time = elapsed_time;
    printf("Time2 = %g ms.\n", elapsed_time);

}

void UseCuda::calMatrixDgemm(const Eigen::MatrixXd &matrix_A, const Eigen::MatrixXd &matrix_B,
                             Eigen::MatrixXd &sol, float &time) {
    float elapsed_time;
    hipEvent_t start1, stop1;
    CHECK(hipEventCreate(&start1));
    CHECK(hipEventCreate(&stop1));
    CHECK(hipEventRecord(start1));
    hipEventQuery(start1);

    typedef double T;
    int M = matrix_A.rows();
    int K = matrix_A.cols();
    int N = matrix_B.cols();
    int MK = M * K;
    int KN = K * N;
    int MN = M * N;

    double *h_A = (double *) malloc(sizeof(double) * MK);
    double *h_B = (double *) malloc(sizeof(double) * KN);
    double *h_C = (double *) malloc(sizeof(double) * MN);

    for (int i = 0; i < K; i++) {
        for (int j = 0; j < M; j++) {
            h_A[i * M + j] = matrix_A(j, i);
        }
    }

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < K; j++) {
            h_B[i * K + j] = matrix_B(j, i);
        }
    }

    for (int i = 0; i < MN; i++) {
        h_C[i] = 0;
    }

    double *g_A, *g_B, *g_C;
    CHECK(hipMalloc((void **) &g_A, sizeof(double) * MK));
    CHECK(hipMalloc((void **) &g_B, sizeof(double) * KN));
    CHECK(hipMalloc((void **) &g_C, sizeof(double) * MN));

    hipblasSetVector(MK, sizeof(double), h_A, 1, g_A, 1);
    hipblasSetVector(KN, sizeof(double), h_B, 1, g_B, 1);
    hipblasSetVector(MN, sizeof(double), h_C, 1, g_C, 1);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double alpha = 1.0;
    double beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                M, N, K, &alpha, g_A, M, g_B, K, &beta, g_C, M);
    hipblasDestroy(handle);

    hipblasGetVector(MN, sizeof(double), g_C, 1, h_C, 1);

    sol = Eigen::MatrixXd::Zero(M, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            sol(j, i) = h_C[i * M + j];
        }
    }
    free(h_A);
    free(h_B);
    free(h_C);
    CHECK(hipFree(g_A));
    CHECK(hipFree(g_B));
    CHECK(hipFree(g_C));

    CHECK(hipEventRecord(stop1));
    CHECK(hipEventSynchronize(stop1));
    CHECK(hipEventElapsedTime(&elapsed_time, start1, stop1));
    time = elapsed_time;
}

void print_matrix(int R, int C, double* A, const char* name)
{
    printf("%s = \n", name);
    for (int r = 0; r < R; ++r)
    {
        for (int c = 0; c < C; ++c)
        {
            printf("%10.6f", A[c * R + r]);
        }
        printf("\n");
    }
}