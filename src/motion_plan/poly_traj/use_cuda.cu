#include "hip/hip_runtime.h"
//
// Created by zhouhua on 2021/4/28.
//

#include "motion_plan/poly_traj/use_cuda.cuh"


template<typename T>
void PrintEMatrix(const T &mat, const char *name) {
    std::cout << name << " =\n";
    std::cout << mat << std::endl;
}
//##############################################################################
template<typename T>
__global__
void Ker_CopyUpperSubmatrix(const T *__restrict d_in,
                            T *__restrict d_ou,
                            const int M, const int N, const int subM) {
    const int i = threadIdx.x + blockIdx.x*blockDim.x;
    const int j = threadIdx.y + blockIdx.y*blockDim.y;
    if (i>=subM || j>=N)
        return;
    d_ou[j*subM+i] = d_in[j*M+i];
}
//##############################################################################

void UseCuda::onInit() {

#define CUSOLVER_ERRCHK(x) \
        assert((x) == HIPSOLVER_STATUS_SUCCESS && "cusolver failed");
#define CUBLAS_ERRCHK(x) \
        assert((x) == HIPBLAS_STATUS_SUCCESS && "cublas failed");
    CUSOLVER_ERRCHK(hipsolverDnCreate(&cusolverH));
    CUBLAS_ERRCHK(hipblasCreate(&cublasH));
}

void UseCuda::calMatrixInverse(const Eigen::MatrixXd &A, Eigen::MatrixXd &x_sol, float &use_time) {
    typedef double T; // NOTE: don't change this. blas has different func name
    typedef Eigen::Matrix<T,Eigen::Dynamic,Eigen::Dynamic> MatrixXd;

    // define handles
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;

    const int M = A.rows();
    const int N = A.cols();

    if (M != N) return;
    const int K = M;

    x_sol.resize(N,K);
//    x_ref = A.inverse();
    MatrixXd b = MatrixXd::Identity(N,K);

    float elapsed_time;

    hipEvent_t start1, stop1;
    CHECK(hipEventCreate(&start1));
    CHECK(hipEventCreate(&stop1));
    CHECK(hipEventRecord(start1));
    hipEventQuery(start1);
/*    PrintEMatrix(A, "A");
    PrintEMatrix(b, "b");
    PrintEMatrix(x_ref, "x_ref");
    PrintEMatrix(C, "C");*/
/*    std::cout << "solution l1 error = " << (x_ref - C).norm()
              << std::endl;*/

#define CUSOLVER_ERRCHK(x) \
        assert((x) == HIPSOLVER_STATUS_SUCCESS && "cusolver failed");
#define CUBLAS_ERRCHK(x) \
        assert((x) == HIPBLAS_STATUS_SUCCESS && "cublas failed");

    CUSOLVER_ERRCHK(hipsolverDnCreate(&cusolverH));
    CUBLAS_ERRCHK(hipblasCreate(&cublasH));



    T *d_A, *d_b, *d_work, *d_work2, *d_tau;
    int *d_devInfo, devInfo;
    CHECK(hipMalloc((void **) &d_A, sizeof(T) * M * N));
    CHECK(hipMalloc((void **) &d_b, sizeof(T) * M * K));
    CHECK(hipMalloc((void **) &d_tau, sizeof(T) * M));
    CHECK(hipMalloc((void **) &d_devInfo, sizeof(int)));
    CHECK(hipMemcpy(d_A, A.data(), sizeof(T) * M * N, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b.data(), sizeof(T) * M * K, hipMemcpyHostToDevice));
    int bufSize, bufSize2;

    // in-place A = QR
    CUSOLVER_ERRCHK(
            hipsolverDnDgeqrf_bufferSize(
                    cusolverH,
                    M,
                    N,
                    d_A,
                    M,
                    &bufSize
            )
    );
    CHECK(hipMalloc((void **) &d_work, sizeof(T) * bufSize));
    CUSOLVER_ERRCHK(
            hipsolverDnDgeqrf(
                    cusolverH,
                    M,
                    N,
                    d_A,
                    M,
                    d_tau,
                    d_work,
                    bufSize,
                    d_devInfo
            )
    );
    CHECK(hipMemcpy(&devInfo, d_devInfo, sizeof(int),
                     hipMemcpyDeviceToHost));
    assert(0 == devInfo && "QR factorization failed");

    // Q^T*b
    CUSOLVER_ERRCHK(
            hipsolverDnDormqr_bufferSize(
                    cusolverH,
                    HIPBLAS_SIDE_LEFT,
                    HIPBLAS_OP_T,
                    M,
                    K,
                    N,
                    d_A,
                    M,
                    d_tau,
                    d_b,
                    M,
                    &bufSize2
            )
    );
    CHECK(hipMalloc((void **) &d_work2, sizeof(T) * bufSize2));
    CUSOLVER_ERRCHK(
            hipsolverDnDormqr(
                    cusolverH,
                    HIPBLAS_SIDE_LEFT,
                    HIPBLAS_OP_T,
                    M,
                    K,
                    min(M, N),
                    d_A,
                    M,
                    d_tau,
                    d_b,
                    M,
                    d_work2,
                    bufSize2,
                    d_devInfo
            )
    );
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(&devInfo, d_devInfo, sizeof(int),
                     hipMemcpyDeviceToHost));
    assert(0 == devInfo && "Q^T b failed");

    // need to explicitly copy submatrix for the triangular solve
    T *d_R, *d_b_;
    CHECK(hipMalloc((void **) &d_R, sizeof(T) * N * N));
    CHECK(hipMalloc((void **) &d_b_, sizeof(T) * N * K));
    dim3 thd_size(10, 10);
    dim3 blk_size((N + thd_size.x - 1) / thd_size.x, (N + thd_size.y - 1) / thd_size.y);
    Ker_CopyUpperSubmatrix<T><<<blk_size, thd_size>>>(d_A, d_R, M, N, N);
    blk_size = dim3((N + thd_size.x - 1) / thd_size.x, (K + thd_size.y - 1) / thd_size.y);
    Ker_CopyUpperSubmatrix<T><<<blk_size, thd_size>>>(d_b, d_b_, M, K, N);

    // solve x = R \ (Q^T*B)
    const double one = 1.0;
    CUBLAS_ERRCHK(
            hipblasDtrsm(
                    cublasH,
                    HIPBLAS_SIDE_LEFT,
                    HIPBLAS_FILL_MODE_UPPER,
                    HIPBLAS_OP_N,
                    HIPBLAS_DIAG_NON_UNIT,
                    N,
                    K,
                    &one,
                    d_R,
                    N,
                    d_b_,
                    N
            )
    );
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(x_sol.data(), d_b_, sizeof(T) * N * K,
                     hipMemcpyDeviceToHost));


    CHECK(hipEventRecord(stop1));
    CHECK(hipEventSynchronize(stop1));
    CHECK(hipEventElapsedTime(&elapsed_time, start1, stop1));
    use_time = elapsed_time;
    printf("Time2 = %g ms.\n", elapsed_time);

/*    PrintEMatrix(x_ref, "x_ref");
    PrintEMatrix(x_sol, "x_sol");*/
}